#include "hip/hip_runtime.h"
#include <stdio.h>      
#include <math.h>    
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include </u/local/cuda/5.0/include/hip/hip_runtime.h>
#include </u/local/cuda/5.0/include/hip/hip_runtime.h>
#include </u/local/cuda/5.0/include/hiprand/hiprand_kernel.h>
#include "./NDR_Solver.cu"
using namespace std;
#define VARIATION
//#define OUTPUT_DETAIL
#define shaodi_pi 3.1415926
#define CUDA_CALL(x) do { if( (x) ! =  hipSuccess ){\
	printf("Error at %s:%d\n",__FILE__,__LINE__ );\
	exit(1);} } while(0) 
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {\
	printf("Error at %s:%d\n",__FILE__,__LINE__);\
	exit(1);}} while(0)

#ifndef OUTPUT_DETAIL
__global__ void LLG(double* g_v_para, double* writeSuccess, int initialstate, double t_step, 
int trials_p_thread, bool isPS, double ori_length, double sigma_l, double ori_width, double sigma_w, 
double ori_tfl, double sigma_tfl, double sigma_mgo, double ori_Nx, double ori_Ny, double ori_Nz, 
double* g_lin_dep_factor, int isNDR, const double* VIGndr, const double* VIGmos, double* g_Energy, 
double* g_SwitchingTime, double* g_EndVndr, double Cload ){
#endif
#ifdef OUTPUT_DETAIL
__global__ void LLG(double* g_v_para, double* writeSuccess, int initialstate, double t_step,
int trials_p_thread, bool isPS, double ori_length, double sigma_l, double ori_width, 
double sigma_w, double ori_tfl, double sigma_tfl, double sigma_mgo, double ori_Nx, 
double ori_Ny, double ori_Nz, double* g_lin_dep_factor, int isNDR, const double* VIGndr, 
const double* VIGmos, double* g_Energy, double* g_SwitchingTime, double* g_EndVndr, double Cload,
double* g_NDRturn, double* g_initialR, double* g_NDRoff){
#endif
/* -------------------------------------------
 Input Parameters From User
 -------------------------------------------*/
int this_id = (blockIdx.x * blockDim.x + threadIdx.x) ;
//initiate state for following random generation
hiprandState_t localState;
hiprand_init(this_id, this_id, 0, &localState);

double Nx = ori_Nx;//origin:                                                // x Demagnetization factor
double Ny = ori_Ny;                                                // y Demagnetization factor
double Nz = ori_Nz;                                                // z Demagnetization factor

//Parameter calculation
double length = ori_length; //length of MTJ
double width = ori_width; //width of MTJ
double Rp0 = 2e3;
double dMgO_a = 1.54e-3, dMgO_b = 1.1537278e10;//origin:9.24e9;
double Area = shaodi_pi*length*width/4;            // Area without variation
double dMgO_base = (log(Rp0 * Area * 10e12) - log(dMgO_a)) / dMgO_b;     // MgO thickness [m]
double TMR = 1.5;                                    // TMR at zero bias voltage
double Rap0 = Rp0 *( 1+TMR);
double Temperature = g_v_para[11];//27+273;                       // Temperature
double pulse_width = g_v_para[1];
double V_p = g_v_para[2], V_ap = g_v_para[3];
double sigma_V_p = g_v_para[4], sigma_V_ap = g_v_para[5];
if( sigma_V_p == 0){
	sigma_V_p = 1e-9;
}
double mean_tr = g_v_para[6], sigma_tr = g_v_para[7], mean_tf = g_v_para[8], sigma_tf= g_v_para[9], delay_time = 0e-9, sense_time = g_v_para[10]; 

int n_sim = (pulse_width+delay_time + sense_time)/t_step ;//Simulation time

//double initial_Vndr = 0;
#ifdef OUTPUT_DETAIL
double	peak_voltage = Peak_voltage(VIGndr); 
double	peak_current = IG_V(peak_voltage,VIGndr,1);
#endif

/* -------------------------------------------
 Constants
 -------------------------------------------*/

double hbar = 1.05457173e-34;                                      // Reduced Planck constant, [J*s]
double k = 1.3806488e-23;                                          // Boltzmann constant, [J/K]
double u0 = 4e-7*shaodi_pi;                                               // Vacuum permeability, [V�s/(A�m)]
double q = 1.60217657e-19;                                         // Electron charge, [C]
double alphac = 0.02;                                              // LLGE damping factor
double gammap = (221276/(1+pow(alphac,2)));                             // Gyromagnetic ratio [m/(A x s)]
double T0 = 1120;
double Ms0 = 1393128.323;//origin:1.44e6;
double Ki0 =1.479036e-3;//origin:1.46e-3;
double Xi0 = 0; //53.39247e-15; //origin:58.9e-15;
if(isPS) Xi0 = 53.39247e-15;
double P_tunnel = 0.2;                                  // the polarization of the tunnel currentdouble
double Pol = 0.6;                                                  // Polarization for Spin Torque

/******************simulation trials *************/
for( int i_trial = 0; i_trial < trials_p_thread; i_trial++){

length = ori_length; //length of MTJ
width = ori_width; //width of MTJ
double tfl = ori_tfl; //thickness of free layer

double rise_time = mean_tr;
double fall_time = mean_tf;
#ifdef VARIATION
//Dimention variation
rise_time += sigma_tr*hiprand_normal_double(&localState);
fall_time += sigma_tf*hiprand_normal_double(&localState);
double v_variation = sigma_V_p * hiprand_normal_double(&localState);
length = ori_length + sigma_l*hiprand_normal_double(&localState); 
width = ori_width + sigma_w*hiprand_normal_double(&localState);
tfl = ori_tfl + sigma_tfl*hiprand_normal_double(&localState);
double dMgO = dMgO_base + sigma_mgo*hiprand_normal_double(&localState);
double temp_Nx = ori_Nx + g_lin_dep_factor[0]*(length-ori_length) + g_lin_dep_factor[1] * ( width - ori_width) + g_lin_dep_factor[2] * (tfl - ori_tfl) ;
double temp_Ny = ori_Ny + g_lin_dep_factor[3]*(length-ori_length) + g_lin_dep_factor[4] * ( width - ori_width) + g_lin_dep_factor[5] * (tfl - ori_tfl) ;
double temp_Nz = ori_Nz + g_lin_dep_factor[6]*(length-ori_length) + g_lin_dep_factor[7] * ( width - ori_width) + g_lin_dep_factor[8] * (tfl - ori_tfl) ;
Nx = temp_Nx / (temp_Nx + temp_Ny + temp_Nz);
Ny = temp_Ny / (temp_Nx + temp_Ny + temp_Nz);
Nz = temp_Nz / (temp_Nx + temp_Ny + temp_Nz); 
#endif

Area = shaodi_pi*length*width/4;            // Area without variation
double areamtj = Area  ;                                // MTJ area [m^2]
double Rp = exp(dMgO * dMgO_b)*dMgO_a / (Area * 10e12);
double Rap = (1+TMR)*Rp;                                           // Anti-parallel resistance [Ohms]
double B1 = 0;//origin 0.2                                   // Field-like torque linear parameter [unitless]
double B2 = 0;//origin 0.02;                                     // Field-like torque quadratic parameter [1/A]
int initial_state = initialstate;                          // Inital state [0 = parallel, 1 = anti-parallel]
double P [3] = {0, 0, -1};                             // Direction of polarization
double Ext [3] = {0, 0, 0};                        // External magnetic field [A/m] - 1 oersted [Oe] = 79.5774715459424 ampere/meter [A/m]


//double t_delay = 2e-9;                             // Time to initiate pulse application [s]

double Ms = Ms0 * ( 1 - pow(Temperature/T0,1.5));                  // Saturation magnetization [A/m] - 1e6 A/m = 1000 emu/cc 
double dstray = 20e-9, tstray = 1.164656e-9;
//double Ext[3]	 = {-Ms*length*width/4/shaodi_pi*((dstray+tstray)/(pow(length/2,2)*sqrt(pow(length/2,2)+pow(dstray+tstray,2)))-(dstray-tstray)/(pow(length/2,2)*sqrt(pow(length/2,2)+pow(dstray-tstray,2)))),0,0};
double Ki = Ki0 * pow(Ms/Ms0, 2.18);                          // Anisotropy field constant [J/m^2]
double Xi = Xi0* pow(Ms/Ms0, 2.83);                                // VCMA field constant [J/(V x m)]
double Gt = 1/(Rp*(1+(TMR/(TMR+2))));                              // Direct elastic tunneling conductance [S]
double KiPF = (2*Ki)/(tfl*u0*Ms);                                  // Prefactor for interface anisotropy effective field
double VCMAPF = (2*Xi)/(u0*Ms*dMgO*tfl);                           // Prefactor for VCMA effective field
double Gsi	= 0;                                                    // Conductance due to imperfections in Mgo [S]

//double Jc0 = (2*Ms*tfl*q*u0)/(hbar*Pol);                           // Normalization Constant for Current Density

double volume = areamtj*tfl;                                       // MTJ volume [m^3]
double Hth = sqrt((2*k*Temperature*alphac)/(u0*gammap*Ms*volume*t_step));    // Amplitude of Thermal Field
//int this_id = (blockIdx.x * blockDim.x + threadIdx.x) * trials_p_thread + i_trial;

/* -------------------------------------------
 Internal Variables
 -------------------------------------------*/

double costheta = 0;                                       // the angle between the magnization  of free and reference layers
double g_sv = 0;                                        // the polarization efficiency in spin valve
double g_tunnel = 0;                                    // the polarization efficiency in tunnel current


//double m_old [3] = {0, 0, 0};                              // Normalized previous magnetization
double Heff_old [3] = {0, 0, 0};                           // Previous Heff components [A/m]
double m_int [3] = {0, 0, 0};                              // Intermediate normalized magnetization
double dm_int [3] = {0, 0, 0};                             // Intermediate derivative of magnetization
double M_int [3] = {0, 0, 0};                              // Intermediate denormalized magnetization                              
//double Heff_int [3] = {0, 0, 0};                           // Intermediate Heff components [A/m]
double dm [3] = {0, 0, 0};                                 // Time derivative of magnetization [1/s]
double M [3] = {0, 0, 0};                                  // Denormalized magnetization
double mcrossp_int [3] = {0, 0, 0};                        // Intermediate cross product components (m x p)
double mcrossHeff_int [3] = {0, 0, 0};                     // Intermediate cross product components (m x Heff)
double mcrossHth_int [3] = {0, 0, 0};                      // Intermediate cross product components (m x Hth)
double mcrossmcrossp_int [3] = {0, 0, 0};                  // Intermediate double cross product components (m x m x p)
double mcrossmcrossHeff_int [3] = {0, 0, 0};               // Intermediate double cross product components (m x m x Heff)
double mcrossp [3] = {0, 0, 0};                            // Cross product components (m x p)
double mcrossHeff [3] = {0, 0, 0};                         // Cross product components (m x Heff)
double mcrossHth [3] = {0, 0, 0};                          // Cross product components (m x Hth)
double mcrossmcrossp [3] = {0, 0, 0};                      // Cross product components (m x m x p)
double mcrossmcrossHeff [3] = {0, 0, 0};                   // Cross product components (m x m x Heff)
double randomHth [3] = {0, 0, 0};                          // Vector of random variables
double STT  = 0;                                        // Strenght of STT term
double FLT  = 0;                                        // Strenght of FLT term

// -------------------------------------------
// Initialize Variables
// -------------------------------------------
double m [3] = {0, 0, 1};                             // Normalized mangetization
double R  = Rap;                						// MTJ resistance [Ohms]
if(initial_state != 1){
                       
    R  = Rp;                                    // MTJ resistance [Ohms]
    m[2]  = -1;                             // Normalized mangetization
}

double J = 0;                                          // Current density [A/m^2]
double V = 0;                                          // MTJ Voltage [V]

 
double V_offset = 0;//1e-10 * (this_id * trials_p_thread+i_trial);

/*********** edition for NDR starts here ***********/
//The parameters for calculating NDR
double Vndr = 0; 
double Imtj = 0; //current through MTJ and nmos
double Vmos = 0; 
double d_Rmtj = 0; // delta Rmtj
double d_Imtj = 0; // delta Imtj
double Indr = 0; //current through NDR
double d_Vndr = 0; //
double Csline = Cload;
double Cbline = Cload;
double d_vdd =0, new_vdd =0, vdd =0;
V = 0; // mtj voltage
#ifdef OUTPUT_DETAIL
	bool isNDRturn = false, isNDRoff = false;
	g_NDRturn [this_id*trials_p_thread + i_trial] = 0;
	g_NDRoff [this_id*trials_p_thread + i_trial] = 0;
	g_initialR [this_id*trials_p_thread + i_trial] = R;
#endif
if(isNDR == 1){ //NDR write
	//Vndr = initial_Vndr;
	Vndr = Solve_stable_vndr(VIGndr, VIGmos, R, V_ap);
	Indr = IG_V(Vndr, VIGndr,1);
	Imtj = Indr;
	Vmos = V_I(Imtj,VIGmos);
	vdd = V_ap;
}
double energy = V_ap*V_ap*Cload; // Pre-charge energy
bool isSwitched = false;
g_SwitchingTime[this_id * trials_p_thread+i_trial] = pulse_width;
/*********** edition for NDR ends here ***********/

for(int i=1;i<=n_sim;i++){

    // Update values
    double m_old [3] = {m[0], m[1], m[2]};
     
    
    // Update voltage/current density
    double V_ub = V_ap + V_offset;
#ifdef VARIATION
    V_ub += v_variation * ( 1 + (sigma_V_ap/sigma_V_p - 1) * (R - Rp0)/(Rap0 - Rp0) ) ;
#endif
    double curr_time = i * t_step;
    if(curr_time < delay_time || curr_time > delay_time + pulse_width){
	new_vdd = 0;
    }
    else{
	if(curr_time < delay_time + rise_time){
		new_vdd = (curr_time - delay_time)/rise_time * V_ub;
	}
	else{
	    if(curr_time <= delay_time + pulse_width - fall_time){
		new_vdd = V_ub;
	    }
	    else{
		new_vdd = V_ub * (  delay_time + pulse_width - curr_time) / fall_time;
	    }
	}
    }
    d_vdd = new_vdd - vdd;
    vdd = new_vdd;
//NDR calculation
/*********** edition for NDR starts here ***********/

#ifdef OUTPUT_DETAIL
    if(isNDR==1){
	if( !isNDRturn && abs(Vndr) > abs(peak_voltage)){
		g_NDRturn [this_id*trials_p_thread + i_trial] = R;
		isNDRturn = true;
	}
	if( isNDRturn && !isNDRoff && abs(Indr) < 0.25*abs(peak_current)){
		g_NDRoff [this_id*trials_p_thread + i_trial] = R;
		isNDRoff = true;
	}
    } else if(isNDR>=2){
	if( !isNDRturn && abs(Vndr) > abs(peak_voltage)){
		g_NDRturn [this_id*trials_p_thread + i_trial] = curr_time;
		isNDRturn = true;
	}
	if (!isNDRoff && curr_time >= delay_time + pulse_width+sense_time){
//	if (!isNDRoff && i>=1){
		isNDRoff = true;
//		g_initialR [this_id*trials_p_thread + i_trial] = d_Vndr;
//		g_NDRturn [this_id*trials_p_thread + i_trial] = Imtj;
	    	g_NDRoff [this_id*trials_p_thread + i_trial] = Vndr + V + Vmos;
	}		
    }
#endif
    if(isNDR==1 ){ //NDR write for AP-MTJ
	//Solve the series of one NDR, one MTJ and one MOS with cap at MTJ
	d_Imtj = ( t_step*Indr - t_step*Imtj - Cload*Imtj*d_Rmtj) / ( Cload/IG_V(Vmos,VIGmos,2) + Cload*R);
	Imtj += d_Imtj;
	Vmos = V_I(Imtj, VIGmos);
	Vndr = vdd - Imtj*R - Vmos;
	Indr = IG_V(Vndr,VIGndr,1);
	V = Imtj*R;
    	energy += vdd * Indr * t_step;
    }
    else if(isNDR == 3){ // NDR read
	if(curr_time < delay_time+pulse_width){ // precharging
            //solve equation: Imtj = (vdd - Vmos(Imtj) - Vndr)/Rmtj = d(Csline*Vndr)/dt + Indr(Vndr)
            Vmos = V_I(Imtj,VIGmos);
            d_Imtj = (d_vdd - d_Vndr - d_Rmtj*Imtj)/(R+1/IG_V(Vmos,VIGmos,2));
            Imtj += d_Imtj;
            Indr = IG_V(Vndr,VIGndr,1);
            d_Vndr = ( Imtj - Indr ) * t_step / Csline;
            Vndr += d_Vndr;
            V = Imtj*R;
    	    energy += vdd * Imtj * t_step;
        }
        else{ // discharging state
            //Solve equation: - d ((Imtj*R + Vmos(Imtj) + Vndr)*Cbline )/dt = Imtj = d(Vndr*Csline)/dt + Indr(Vndr)

            d_Imtj = (-Imtj*t_step - Cbline*Imtj*d_Rmtj -d_Vndr*Cbline) / ( Cbline/IG_V(Vmos,VIGmos,2) + Cbline*R);
            Imtj += d_Imtj;
            Vmos = V_I(Imtj,VIGmos);
            d_Vndr = (Imtj - Indr) * t_step / Csline;
            Vndr += d_Vndr;
            Indr = IG_V(Vndr,VIGndr,1);
            V = Imtj*R;
        }
    }
    else if (isNDR == 2 || isNDR == 0){
	 if(curr_time < delay_time+pulse_width){ // precharging
            //solve equation: Imtj = (vdd - Vmos(Imtj) )/Rmtj = 0
            d_Imtj = (d_vdd-d_Rmtj*Imtj)/(R+1/IG_V(Vmos,VIGmos,2));
            Imtj += d_Imtj;
            V = Imtj*R;
            Vmos = vdd - V;
	    Indr = Imtj + R*Cbline*d_Imtj/t_step + Imtj*Cbline*d_Rmtj/t_step + Cbline/IG_V(Vmos,VIGmos,2)*d_Imtj/t_step;//The total I but not current of ndr, because there is no ndr
	    energy += vdd * Indr * t_step;
        }
        else{ // discharging state
            //Solve equation: - d ((Imtj*R + Vmos(Imtj) )*Cbline )/dt = Imtj 
    
            d_Imtj = (-Imtj*t_step - Cbline*Imtj*d_Rmtj ) / ( Cbline/IG_V(Vmos,VIGmos,2) + Cbline*R);
            Imtj += d_Imtj;
            Vmos = V_I(Imtj,VIGmos);
            V = Imtj*R;
        }
    }
//    else{
	//Solve the series of one MTJ and one MOS with cap at MTJ
//	if(curr_time < delay_time+pulse_width){
//	    d_Imtj = (d_vdd-d_Rmtj*Imtj)/(R+1/IG_V(Vmos,VIGmos,2));
//            Imtj += d_Imtj;
//            V = Imtj*R;
//            Vmos = vdd - V;
//	    Indr = Imtj + R*Cload*d_Imtj/t_step + Imtj*Cload*d_Rmtj/t_step + Cload/IG_V(Vmos,VIGmos,2)*d_Imtj/t_step;//The total I but not current of ndr, because there is no ndr
//	    energy += vdd * Indr * t_step;
//    }
//Test whether switched
    if(!isSwitched){
	if( (initial_state ==0 && R >= Rp*(1+TMR/2)) || ( initial_state ==1 && R <= Rp*(1+TMR/2)) ){
	    isSwitched = true;
	    g_SwitchingTime[this_id * trials_p_thread+i_trial] = curr_time;
	}
    }
            
/*********** edition for NDR ends here ***********/

    
	

    // Update effective magnetic field Heff_old
    Heff_old[0] = Ext[0]-Ms*Nx*m_old[0];
    Heff_old[1] = Ext[1]-Ms*Ny*m_old[1];
    Heff_old[2] = Ext[2]-Ms*Nz*m_old[2]+(KiPF*m_old[2]-VCMAPF*m_old[2]*V);
    
    //Calculate STT factor
    J = V/(R*areamtj);
//    costheta = m_old[0]*P[0] + m_old[1]*P[1] + m_old[2]*P[2];
//    g_tunnel = 1/2 * P_tunnel / ( 1 + pow(P_tunnel,2)*costheta);
//    g_sv = 1 / ( -4 + pow(( 1 / sqrt(Pol) + sqrt(Pol) ), 3) * (3 + costheta) / 4);
//    STT = gammap*J* hbar*(g_tunnel+g_sv)/(2*Ms*tfl*q*u0);
    STT = Pol*gammap*J* hbar/(2*Ms*tfl*q*u0);
    //STT = gammap*J/Jc0;
    FLT = STT*B1+STT*B2*areamtj*J;

    // Calculate m x Hth
    mcrossHth_int[0]=m_old[1]*randomHth[2]-m_old[2]*randomHth[1];
    mcrossHth_int[1]=m_old[2]*randomHth[0]-m_old[0]*randomHth[2];
    mcrossHth_int[2]=m_old[0]*randomHth[1]-m_old[1]*randomHth[0];

    // Calculate m x p and m x m x p
    mcrossp_int[0]=m_old[1]*P[2]-m_old[2]*P[1];
    mcrossp_int[1]=m_old[2]*P[0]-m_old[0]*P[2];
    mcrossp_int[2]=m_old[0]*P[1]-m_old[1]*P[0];
    mcrossmcrossp_int[0]=m_old[1]*mcrossp_int[2]-m_old[2]*mcrossp_int[1];
    mcrossmcrossp_int[1]=m_old[2]*mcrossp_int[0]-m_old[0]*mcrossp_int[2];
    mcrossmcrossp_int[2]=m_old[0]*mcrossp_int[1]-m_old[1]*mcrossp_int[0];

    // Calculate m x Heff and m x m x Heff
    mcrossHeff_int[0]=m_old[1]*Heff_old[2]-m_old[2]*Heff_old[1];
    mcrossHeff_int[1]=m_old[2]*Heff_old[0]-m_old[0]*Heff_old[2];
    mcrossHeff_int[2]=m_old[0]*Heff_old[1]-m_old[1]*Heff_old[0];
    mcrossmcrossHeff_int[0]=m_old[1]*mcrossHeff_int[2]-m_old[2]*mcrossHeff_int[1];
    mcrossmcrossHeff_int[1]=m_old[2]*mcrossHeff_int[0]-m_old[0]*mcrossHeff_int[2];
    mcrossmcrossHeff_int[2]=m_old[0]*mcrossHeff_int[1]-m_old[1]*mcrossHeff_int[0];
    // Use the LLG equation w/ Heun's Method to update the magnetization
    dm_int[0] = -gammap*(mcrossHeff_int[0]+mcrossHth_int[0]) - gammap*alphac*mcrossmcrossHeff_int[0] + STT*mcrossmcrossp_int[0] + FLT*mcrossp_int[0];
    dm_int[1] = -gammap*(mcrossHeff_int[1]+mcrossHth_int[1]) - gammap*alphac*mcrossmcrossHeff_int[1] + STT*mcrossmcrossp_int[1] + FLT*mcrossp_int[1];
    dm_int[2] = -gammap*(mcrossHeff_int[2]+mcrossHth_int[2]) - gammap*alphac*mcrossmcrossHeff_int[2] + STT*mcrossmcrossp_int[2] + FLT*mcrossp_int[2];
    M_int[0] = m_old[0] + (dm_int[0]*t_step);
    M_int[1] = m_old[1] + (dm_int[1]*t_step);
    M_int[2] = m_old[2] + (dm_int[2]*t_step);
    m_int[0] = M_int[0]/sqrt(M_int[0]*M_int[0]+M_int[1]*M_int[1]+M_int[2]*M_int[2]);
    m_int[1] = M_int[1]/sqrt(M_int[0]*M_int[0]+M_int[1]*M_int[1]+M_int[2]*M_int[2]);
    m_int[2] = M_int[2]/sqrt(M_int[0]*M_int[0]+M_int[1]*M_int[1]+M_int[2]*M_int[2]);

    // Update the thermal field and current values (time evolves)
    
	double2 gen_x12;
	double gen_x3;
	gen_x12 = hiprand_normal2_double(&localState);
      randomHth[0] = Hth*gen_x12.x;
      randomHth[1] = Hth*gen_x12.y;
	gen_x3 = hiprand_normal_double(&localState);
      randomHth[2] = Hth*gen_x3;
    
//STT calculation
//    costheta = m_int[0]*P[0] + m_int[1]*P[1] + m_int[2]*P[2];
//    g_tunnel = 1/2 * P_tunnel / ( 1 + pow(P_tunnel,2)*costheta);
//    g_sv = 1 / ( -4 + pow(( 1 / sqrt(Pol) + sqrt(Pol) ), 3) * (3 + costheta) / 4); 
//    STT = gammap*J* hbar*(g_tunnel+g_sv)/(2*Ms*tfl*q*u0);
    STT = Pol*gammap*J* hbar/(2*Ms*tfl*q*u0);

    //STT = gammap*J/Jc0;
    FLT = STT*B1+STT*B2*areamtj*J;

    // Update intermediate effective magnetic field Heff
    double Heff_int [3] = {Ext[0]-Ms*Nx*m_int[0], Ext[1]-Ms*Ny*m_int[1], Ext[2]-Ms*Nz*m_int[2]+(KiPF*m_int[2]-VCMAPF*m_int[2]*V)};

    // Calculate m x Hth
    mcrossHth[0]=m_int[1]*randomHth[2]-m_int[2]*randomHth[1];
    mcrossHth[1]=m_int[2]*randomHth[0]-m_int[0]*randomHth[2];
    mcrossHth[2]=m_int[0]*randomHth[1]-m_int[1]*randomHth[0];
    // Calculate m x p and m x m x p
    mcrossp[0]=m_int[1]*P[2]-m_int[2]*P[1];
    mcrossp[1]=m_int[2]*P[0]-m_int[0]*P[2];
    mcrossp[2]=m_int[0]*P[1]-m_int[1]*P[0];
    mcrossmcrossp[0]=m_int[1]*mcrossp[2]-m_int[2]*mcrossp[1];
    mcrossmcrossp[1]=m_int[2]*mcrossp[0]-m_int[0]*mcrossp[2];
    mcrossmcrossp[2]=m_int[0]*mcrossp[1]-m_int[1]*mcrossp[0];

    // Calculate m x Heff and m x m x Heff
    mcrossHeff[0]=m_int[1]*Heff_int[2]-m_int[2]*Heff_int[1];
    mcrossHeff[1]=m_int[2]*Heff_int[0]-m_int[0]*Heff_int[2];
    mcrossHeff[2]=m_int[0]*Heff_int[1]-m_int[1]*Heff_int[0];
    mcrossmcrossHeff[0]=m_int[1]*mcrossHeff[2]-m_int[2]*mcrossHeff[1];
    mcrossmcrossHeff[1]=m_int[2]*mcrossHeff[0]-m_int[0]*mcrossHeff[2];
    mcrossmcrossHeff[2]=m_int[0]*mcrossHeff[1]-m_int[1]*mcrossHeff[0];

    // Now use intermediate value in final value computation 
    dm[0] = -gammap*(mcrossHeff[0]+mcrossHth[0]) - gammap*alphac*mcrossmcrossHeff[0] + STT*mcrossmcrossp[0] + FLT*mcrossp[0];
    dm[1] = -gammap*(mcrossHeff[1]+mcrossHth[1]) - gammap*alphac*mcrossmcrossHeff[1] + STT*mcrossmcrossp[1] + FLT*mcrossp[1];
    dm[2] = -gammap*(mcrossHeff[2]+mcrossHth[2]) - gammap*alphac*mcrossmcrossHeff[2] + STT*mcrossmcrossp[2] + FLT*mcrossp[2];
    M[0] = m_old[0] + (t_step/2)*(dm[0] + dm_int[0]);
    M[1] = m_old[1] + (t_step/2)*(dm[1] + dm_int[1]);
    M[2] = m_old[2] + (t_step/2)*(dm[2] + dm_int[2]);
    m[0] = M[0]/sqrt(M[0]*M[0]+M[1]*M[1]+M[2]*M[2]);
    m[1] = M[1]/sqrt(M[0]*M[0]+M[1]*M[1]+M[2]*M[2]);
    m[2] = M[2]/sqrt(M[0]*M[0]+M[1]*M[1]+M[2]*M[2]);
    // Update final values for next step

/*********** edition for NDR starts here ***********/
    d_Rmtj = 1/(Gt*(1+(TMR/(TMR+2))*(m[0]*P[0]+m[1]*P[1]+m[2]*P[2]))+Gsi) - R;
/*********** edition for NDR ends here ***********/
    R = 1/(Gt*(1+(TMR/(TMR+2))*(m[0]*P[0]+m[1]*P[1]+m[2]*P[2]))+Gsi);


}
   	if( initial_state ==0){
   		if( R >= Rp*(1+TMR/2)){
   			writeSuccess[this_id*trials_p_thread + i_trial]=1;
   		}
		else{
			writeSuccess[this_id*trials_p_thread + i_trial]=0;
		}
   	}
   	else {
   		if( R <= Rp*(1+TMR/2)){
   			writeSuccess[this_id*trials_p_thread + i_trial]=1;
   		}
		else{
			writeSuccess[this_id*trials_p_thread + i_trial]=0;
		}
   	}

//Recording switching energy and voltage after switching
	g_Energy[this_id * trials_p_thread+i_trial] = energy;
	if(isNDR <=1){
		g_EndVndr[this_id * trials_p_thread+i_trial] = Vndr;
	}
	else{
		g_EndVndr[this_id * trials_p_thread+i_trial] = Vndr+Vmos+V;
	}

}
                  
}


