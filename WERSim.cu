#include "hip/hip_runtime.h"
// -------------------------------------------
// Precessional Switching Old Parameters
// Updated November 21 - 2013
// Vector coordinates: [x; y; z]
// -------------------------------------------
#include <iostream>
#include <math.h>
#include "LLG.cu"
#include <vector>
#include <fstream>
#include <iomanip>
using namespace std;
// -------------------------------------------
// Calculate Wall time
// -------------------------------------------


//#define DEBUG

#include <sys/time.h>
#include "./Demagnetization_factors.cu"
double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}



// -------------------------------------------
// WER Calculation
// -------------------------------------------
static hipError_t crc;

void g_allocate_1D(double **g_f, int nsize, int *irc) {
/* allocate global double memory on GPU, return pointer to C */
   void *gptr;
   crc = hipMalloc(&gptr,sizeof(double)*nsize);
   if (crc) {
      printf("hipMalloc double Error=%d:%s,l=%d\n",crc,
              hipGetErrorString(crc),nsize);
      *irc = 1;
   }
   *g_f = (double *)gptr;
   return;
}

void g_allocate_2D(double ***g_f, size_t * pitch, int row, int column, int *irc) {
/* allocate global double memory on GPU, return pointer to C */
   void *gptr;
   crc = hipMallocPitch( &gptr, pitch, sizeof(double)*column, row);
   if (crc) {
      printf("hipMalloc double Error=%d:%s,row=%d,column=%d\n",crc,
              hipGetErrorString(crc),row,column);
      *irc = 1;
   }
   *g_f = (double **)gptr;
   return;
}


void copyin_gmemptr_1D (double *f, double *g_f, int nsize) {
/* copy double array from main memory to global GPU memory */
   crc = hipMemcpy((void *)g_f,f,sizeof(double)*nsize,
                    hipMemcpyHostToDevice);
   if (crc) {
      printf("cudaMemcpyHostToDevice1D double Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

void copyin_gmemptr_2D (double **f, double **g_f, size_t& pitch, int row, int column) {
/* copy double array from main memory to global GPU memory */
   crc = hipMemcpy2D(g_f,pitch,f,sizeof(double)*column,sizeof(double)*column, row, hipMemcpyHostToDevice);
   if (crc) {
      printf("cudaMemcpyHostToDevice2D double Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

void copyout_gmemptr_1D(double *f, double *g_f, int nsize) {
/* copy double array from global GPU memory to main memory */
   crc = hipMemcpy(f,g_f,sizeof(double)*nsize,
                    hipMemcpyDeviceToHost);
   if (crc) {
      printf("cudaMemcpyDeviceToHost1D double Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

void copyout_gmemptr_2D(double **f, double **g_f, size_t &pitch, int row, int column) {
/* copy double array from main memory to global GPU memory */
   crc = hipMemcpy2D(f,sizeof(double)*column,g_f,pitch,sizeof(double)*column, row, hipMemcpyDeviceToHost);
   if (crc) {
      printf("cudaMemcpyDeviceToHost2D double Error=%d:%s\n",crc,
              hipGetErrorString(crc));
      exit(1);
   }
   return;
}

int main(int argc, char* argv[])
{
	if(argc < 11) {
		cout<<" arguments: [trials] [number of blocks (32n)] [number of threads in block (32m)]  [initial state 0 = parallel, 1 = anti-parallel] [input pulse shape file] [enable VCMA?, 1:enable, 0: disable] [v_i characteristics of NDR] [v_i characteristics of MOS] [ndr write/read? 0: ndr write without ndr, 1: ndr write, 2: normal read without ndr, 3: ndr read; rise time is required except option 1] [Cbitline]"<<endl;
	return 1;
	}
	//dimention 
        double length = 50e-9;                            // MTJ length [m]
	double width = 50e-9;                            // MTJ width [m]
	double tfl = 1.18e-9;//origin:1.1e-9                               // Free layer thickness [m]
	double sigma_l = 1e-9;
	double sigma_w = 1e-9;
	double sigma_tfl = 0.003e-9;
	double sigma_mgo = 0.003e-9;
	double* lin_dep_factor = new double[9];
	double Cload = atof(argv[10]);
	//Demagnetization calculation
	cout<<"Start Demagnitization calculation"<<endl;
	double Nx = 0, Ny = 0, Nz = 0;
	Extract_linear_dependent(length, sigma_l,  width, sigma_w, tfl, sigma_tfl, Nx, Ny, Nz,lin_dep_factor);

#ifdef DEBUG
	std::cout<<Nx<<" "<<Ny<<" "<<Nz<<" "<<endl;
#endif
		
	double * g_lin_dep_factor;

	int irc = 0;
	g_allocate_1D(&g_lin_dep_factor, 9 ,&irc);
	if(irc!=0){
		cout<<"error in allocating memory in GPU"<<endl;
		return 1;
	}
	copyin_gmemptr_1D(  lin_dep_factor , g_lin_dep_factor, 9);
	
	fstream fs,fi;
	fi.open(argv[5],std::fstream::in);
	vector<double> v_par;
	double readBuffer,voltage=0;
	while(fi >> readBuffer){
		v_par.push_back(readBuffer);
	}
	fi.close();
	voltage = v_par[0]; 
	//Move pulse shape to 1D array to copy to GPU
	double *v_para;
	unsigned int n_par = v_par.size();
	v_para = new double[n_par];
	for (int i_p = 0; i_p < n_par; i_p ++ ){
		v_para[i_p] = v_par[i_p];
	}
	double t_pulse = v_par[1];
	vector<double>().swap(v_par);
	
	double t_step = 3e-12;                             // Time step [s]
	
	int trials = atoi(argv[1]);
	int GridSize = atoi(argv[2]);
	int GridLength = sqrt(GridSize);
	int BlockSize = atoi(argv[3]);
	bool enableVCMA=true; // is it precessional switching or STT
	if ( atoi(argv[6]) == 0){
		 enableVCMA = false;
	}
	
	int BlockLength = sqrt(BlockSize);
	int trials_p_thread = ceil(double(trials)/ double(GridSize*BlockSize));
	int real_trials = trials_p_thread * GridSize*BlockSize;
	cout<< "real number of trials is: "<<real_trials<<endl;
	dim3 dimBlock(BlockSize, 1);
	dim3 dimGrid(GridSize,1);
	int initial_state = atoi(argv[4]);
	if( initial_state == 1){
		fs.open("ap2p.txt",std::fstream::out | std::fstream::app);
	}
	else{
		fs.open("p2ap.txt", std::fstream::out | std::fstream::app);
	}
//Setup random variable

	
	double start_t = get_wall_time();
	cout<<"Start copy from host memory to GPU..."<<endl;

	double* global_writeSuccess = new double[real_trials];
	double * g_writeSuccess, *g_v_para;
	g_allocate_1D(&g_writeSuccess, real_trials ,&irc);
	copyin_gmemptr_1D( global_writeSuccess ,g_writeSuccess, real_trials);
	g_allocate_1D(&g_v_para, n_par, &irc);
	if(irc!=0){
		cout<<"error in allocating memory in GPU"<<endl;
		return 1;
	}
	copyin_gmemptr_1D( v_para, g_v_para, n_par);

/***************************************/
//Edition for NDR starts here	
	double* global_Energy = new double[real_trials];
	double* global_SwitchingTime = new double[real_trials];
	double* global_EndVndr = new double[real_trials];
	double *g_Energy, *g_SwitchingTime, *g_EndVndr, *VIGndr, *VIGmos, *g_VIGndr, *g_VIGmos; // the v and I characteristics of NDR and MOSFET
	int isNDR = atoi(argv[9]); // is NDR is added to the circuit
	int npoint_ndr = N_point; 
	VIGndr = Read_voltage_current(argv[7]); // Read in v_i characteristics
	VIGmos = Read_voltage_current(argv[8]);
//	size_t this_pitch;
//	g_allocate_2D(&g_VIGndr, &this_pitch, npoint_ndr, 3, &irc);
	g_allocate_1D(&g_VIGndr, npoint_ndr*3, &irc);
	g_allocate_1D(&g_VIGmos, npoint_ndr*3, &irc);
	if(irc!=0){
		cout<<"error in allocating memory in GPU"<<endl;
		return 1;
	}
	copyin_gmemptr_1D( VIGndr, g_VIGndr, npoint_ndr*3);
	copyin_gmemptr_1D( VIGmos, g_VIGmos, npoint_ndr*3);
	g_allocate_1D(&g_Energy, real_trials ,&irc);
	g_allocate_1D(&g_SwitchingTime, real_trials ,&irc);
	g_allocate_1D(&g_EndVndr, real_trials ,&irc);
	if(irc!=0){
		cout<<"error in allocating memory in GPU"<<endl;
		return 1;
	}
	copyin_gmemptr_1D( global_Energy ,g_Energy, real_trials);
	copyin_gmemptr_1D( global_SwitchingTime ,g_SwitchingTime, real_trials);
	copyin_gmemptr_1D( global_EndVndr ,g_EndVndr, real_trials);

#ifdef OUTPUT_DETAIL	
	double* global_NDRturnR = new double[real_trials];
	double* global_initialR = new double[real_trials];
	double* global_NDRoffR = new double[real_trials];
	double *g_NDRturnR, *g_initialR, *g_NDRoffR ;
	g_allocate_1D(&g_NDRturnR, real_trials ,&irc);
	g_allocate_1D(&g_initialR, real_trials ,&irc);
	g_allocate_1D(&g_NDRoffR, real_trials ,&irc);
	if(irc!=0){
		cout<<"error in allocating memory in GPU"<<endl;
		return 1;
	}
	copyin_gmemptr_1D( global_NDRturnR ,g_NDRturnR, real_trials);
	copyin_gmemptr_1D( global_initialR ,g_initialR, real_trials);
	copyin_gmemptr_1D( global_NDRoffR ,g_NDRoffR, real_trials);
#endif
// Editional for NDR ends here	
/***************************************/

	double copyin_t = get_wall_time();
	cout<<"Start GPU calculation..."<<endl;
#ifdef OUTPUT_DETAIL
   	LLG<<<dimGrid,dimBlock>>>(g_v_para, g_writeSuccess, initial_state, t_step, trials_p_thread, enableVCMA, length, sigma_l,  width, sigma_w, tfl, sigma_tfl, sigma_mgo, Nx, Ny, Nz, g_lin_dep_factor,isNDR,g_VIGndr,g_VIGmos,  g_Energy, g_SwitchingTime, g_EndVndr, Cload, g_NDRturnR, g_initialR,g_NDRoffR);
#endif
#ifndef OUTPUT_DETAIL
   	LLG<<<dimGrid,dimBlock>>>(g_v_para, g_writeSuccess, initial_state, t_step, trials_p_thread, enableVCMA, length, sigma_l,  width, sigma_w, tfl, sigma_tfl, sigma_mgo, Nx, Ny, Nz, g_lin_dep_factor,isNDR,g_VIGndr,g_VIGmos,  g_Energy, g_SwitchingTime, g_EndVndr,Cload);
#endif
	hipDeviceSynchronize();
	double calculate_t = get_wall_time();
	cout<<"Start copy GPU to host memory..."<<endl;
	copyout_gmemptr_1D(global_writeSuccess, g_writeSuccess, real_trials);
/****************************/
//Edition for NDR starts here
	copyout_gmemptr_1D(global_Energy, g_Energy, real_trials);
	copyout_gmemptr_1D(global_SwitchingTime, g_SwitchingTime, real_trials);
	copyout_gmemptr_1D(global_EndVndr, g_EndVndr, real_trials);
	cout<<"Start couting final result...\n**********************"<<endl;
#ifdef OUTPUT_DETAIL
	copyout_gmemptr_1D(global_NDRturnR, g_NDRturnR, real_trials);
	copyout_gmemptr_1D(global_initialR, g_initialR, real_trials);
	copyout_gmemptr_1D(global_NDRoffR, g_NDRoffR, real_trials);
	fstream fout;
	fout.open("output_detailR.txt",std::fstream::out);
	for (int i_trial = 0; i_trial<real_trials ; i_trial++){
		fout << "switched/R_initial/NDRturn/NDRoff\t"<<global_writeSuccess[i_trial]<<"\t"<<global_initialR[i_trial]<<"\t"<<global_NDRturnR[i_trial]<<"\t"<<global_NDRoffR[i_trial]<<std::endl;
	}
	fout.close();
	
#endif
	double total_energy = 0;
	double total_switchingtime = 0;
	double ave_vndr = 0;
	int read_failures = 0;
	double min_margin = 0;
	if(n_par > 12) {
		min_margin = v_para[12];
	}
	for(int i_e = 0; i_e < real_trials; i_e++){
		total_energy += global_Energy[i_e];
		total_switchingtime += global_SwitchingTime[i_e];
		ave_vndr += global_EndVndr[i_e]; // Sensed voltage
		read_failures += (global_EndVndr[i_e] > min_margin)? 0:1;
	}
	ave_vndr /= real_trials;
	if(isNDR >=2){ // in read mode
		cout<<"Results of read mode ("<<isNDR<<")"<<endl;
		double std_vndr = 0;
		for( int i_trial = 0; i_trial < real_trials; i_trial++){
			std_vndr += (global_EndVndr[i_trial] - ave_vndr)*(global_EndVndr[i_trial] - ave_vndr);
		}
		std_vndr = sqrt(std_vndr/real_trials);
		cout<<"  Average sensing margin is: "<<ave_vndr<<endl;
		cout<<"  Standard deviation of sensing margin is: " << std_vndr<<endl;
		cout<<"  Number of read failures (< minimum margin of "<<min_margin<<"): "<<read_failures<<endl;
	}else{
		cout<<"Results of write mode ("<<isNDR<<")"<<endl;
		cout<<"  Average switching power is: "<< total_energy/real_trials <<"\nAverage switching time is: "<<total_switchingtime/real_trials<<endl;
		cout<<"  Average Vndr after switching is: "<<ave_vndr<<endl;
	}

//Edition for NDR ends here
/***************************/
	double copyout_t = get_wall_time();

	int sum=0;
   	for( int i =0 ; i< real_trials; i++){
   		sum+= global_writeSuccess[i];
   	}
	double couting_t = get_wall_time();
	cout << "**********************\nMonte-Carlo Simulation Results:\n  switching from "<< ((initial_state==0)? "P" : "AP")<<" to "<< ((initial_state==1)? "P" : "AP")<<"\n"
	<<"  Pulse voltage: "<<voltage<<" V, time step: "<<t_step<<"s\n  "
        <<sum<<" trials success out of total "<<real_trials<<" trials, switching rate: "<<std::setprecision(9)<<(double(sum) / double(real_trials)) << endl;
	cout<<"**********************\nRuntime summary:\n"<<" copy memory to GPU: "<<copyin_t - start_t
<<" s, GPU calculation: "<<calculate_t - copyin_t<<" s, copy memory out to CPU: "
<<copyout_t - calculate_t <<" s, couting switching: "<< couting_t - copyout_t <<" s."<<endl;
	fs << voltage <<" "<<t_pulse<<" "<<std::setprecision(9)<<(double(sum) / double(real_trials))<<" "<<couting_t-start_t<<" s"<<endl;
	fs.close();
	return 0;
}





